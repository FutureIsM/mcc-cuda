#include "hip/hip_runtime.h"
#include <vector>

#include "errors.h"
#include "debug.h"
#include "minutia.cuh"
#include "constants.cuh"

using namespace std;

bool initialized = false;
int numCellsInCylinder = 0;

__host__ void initialize() {
  if (initialized) return;

  numCellsInCylinder = 0;
  float temp = DELTA_S/2;
  for (int i = 0; i < NS; ++i) {
    float x = DELTA_S * i + temp;
    float dx = x-R;
    for (int j = 0; j < NS; ++j) {
      float y = DELTA_S * j + temp;
      float dy = y-R;
      if (dx*dx + dy*dy <= R_SQR) ++numCellsInCylinder;
    }
  }
  initialized = true;
}

__host__ __device__ __inline__
float gaussian(float value) {
  const float GS_DIV = sqrtf(M_2PI) * SIGMA_S;
  return expf(-(value*value)/SIGMA_2S_SQR) / GS_DIV;
}

// http://www.wolframalpha.com/input/?i=integrate+(e%5E(-(t%5E2)%2F(2(x%5E2)))+dt)
__host__ __device__ __inline__
float gaussianIntegral(float value) {
  const float a = sqrtf(M_PI_2) * SIGMA_D;
  const float b = M_SQRT2 * SIGMA_D;
  auto integrate = [&](float val) {
    return a * erff(val/b);
  };
  return rsqrtf(M_2PI) *
    (integrate(value+DELTA_D_2)-integrate(value-DELTA_D_2))
    / SIGMA_D;
}

__host__ __device__ __inline__
int sqrDistance(int x1, int y1, int x2, int y2) {
  int dx = x1 - x2;
  int dy = y1 - y2;
  return dx*dx + dy*dy;
}

__host__ __device__ __inline__
float distance(int x1, int y1, int x2, int y2) {
  return sqrtf(sqrDistance(x1, y1, x2, y2));
}

__host__ __device__ __inline__
float spatialContribution(
    int mt_x, int mt_y, int pi, int pj) {
  return gaussian(distance(mt_x, mt_y, pi, pj));
}

__host__ __device__ __inline__
float angle(float theta1, float theta2) {
  float diff = theta1-theta2;
  if (diff < -M_PI)
    return M_2PI + diff;
  if (diff >= M_PI)
    return -M_2PI + diff;
  return diff;
}

__host__ __device__ __inline__
float directionalContribution(
    float m_theta, float mt_theta, float dphik) {
  return gaussianIntegral(
    angle(dphik, angle(m_theta, mt_theta)));
}

__global__
void buildCylinder(
    Minutia *minutiae, char *validArea,
    int width, int height,
    char *cylinderValidities,
    char *cellValues,
    char *cellValidities,
    int numCellsInCylinder) {
  extern __shared__ int shared[];

  const int N = gridDim.x;
  Minutia *sharedMinutiae = (Minutia*)shared;

  int idxMinutia = blockIdx.x;
  int idxThread = threadIdx.y * blockDim.x + threadIdx.x;
  int contributed = 0;

  if (idxThread < N) {
    sharedMinutiae[idxThread] = minutiae[idxThread];
    if (idxThread != idxMinutia) {
      auto dist = sqrDistance(
        sharedMinutiae[idxThread].x, sharedMinutiae[idxThread].y,
        minutiae[idxMinutia].x, minutiae[idxMinutia].y);
      contributed = dist <= (R+SIGMA_3S)*(R+SIGMA_3S);
    }
  }
  int sumContributed = __syncthreads_count(contributed);

  Minutia m = sharedMinutiae[idxMinutia];

  float halfNS = (NS + 1) / 2.0f;
  float halfNSi = (threadIdx.x+1) - halfNS;
  float halfNSj = (threadIdx.y+1) - halfNS;
  float sint, cost;
  sincosf(m.theta, &sint, &cost);
  int pi = m.x + DELTA_S * (cost * halfNSi + sint * halfNSj);
  int pj = m.y + DELTA_S * (-sint * halfNSi + cost * halfNSj);

  char validity = pi >= 0 && pi < width && pj >= 0 && pj < height
    && validArea[pj * width + pi]
    && sqrDistance(m.x, m.y, pi, pj) <= R_SQR;

  int idx = idxMinutia * NC + threadIdx.x * NS * NS + threadIdx.y * NS;
  for (int k = 0; k < ND; ++k, ++idx) {
    char value = 0;

    if (validity) {
      float dphik = -M_PI + (k + 0.5f) * DELTA_D;
      float sum = 0.0f;

      for (int l = 0; l < N; ++l) {
        if (l == idxMinutia)
          continue;

        Minutia mt = sharedMinutiae[l];
        if (sqrDistance(m.x, m.y, mt.x, mt.y) > SIGMA_9S_SQR)
          continue;

        float sContrib = spatialContribution(mt.x, mt.y, pi, pj);
        float dContrib = directionalContribution(m.theta, mt.theta, dphik);
        sum += sContrib * dContrib;
      }

      if (sum >= MU_PSI)
        value = 1;
    }
    cellValidities[idx] = validity;
    cellValues[idx] = value;
  }

  int sumValidities = __syncthreads_count(validity);
  if (threadIdx.x == 0 && threadIdx.y == 0) {
    cylinderValidities[idxMinutia] = sumContributed >= MIN_M &&
      (float)sumValidities/(numCellsInCylinder) >= MIN_VC;
    devDebug("Minutia %d:\nMIN_VC ((%d/%d) = %f) >= %f\nMIN_M %d >= %d\n\n",
      idxMinutia,
      sumValidities, numCellsInCylinder,
      (float)sumValidities/(numCellsInCylinder), MIN_VC,
      sumContributed, MIN_M);
  }
}

__host__
void buildTemplate(
    const vector<Minutia>& minutiae,
    const vector<char>& validArea,
    int width, int height,
    vector<char>& cylinderValidities,
    vector<char>& cellValues,
    vector<char>& cellValidities) {

  initialize();

  Minutia *devMinutiae;
  char *devArea;
  char *devCylinderValidities, *devCellValues, *devCellValidities;
  size_t devMinutiaeSize = minutiae.size() * sizeof(Minutia);
  size_t devAreaSize = width * height * sizeof(char);
  size_t devCylinderValiditiesSize = minutiae.size() * sizeof(char);
  size_t devCellValuesSize = minutiae.size() * NC * sizeof(char);
  size_t devCellValiditiesSize = minutiae.size() * NC * sizeof(char);
  handleError(
    hipMalloc(&devMinutiae, devMinutiaeSize));
  handleError(
    hipMemcpy(devMinutiae, minutiae.data(), devMinutiaeSize, hipMemcpyHostToDevice));
  handleError(
    hipMalloc(&devArea, devAreaSize));
  handleError(
    hipMemcpy(devArea, validArea.data(), devAreaSize, hipMemcpyHostToDevice));
  handleError(
    hipMalloc(&devCylinderValidities, devCylinderValiditiesSize));
  handleError(
    hipMalloc(&devCellValues, devCellValuesSize));
  handleError(
    hipMalloc(&devCellValidities, devCellValiditiesSize));

  dim3 blockDim(NS, NS);
  int sharedSize = devMinutiaeSize + MAX_MINUTIAE * sizeof(char);
  buildCylinder<<<minutiae.size(), blockDim, sharedSize>>>(
    devMinutiae, devArea, width, height,
    devCylinderValidities, devCellValues, devCellValidities,
    numCellsInCylinder);

  cylinderValidities.resize(minutiae.size());
  cellValues.resize(minutiae.size() * NC);
  cellValidities.resize(minutiae.size() * NC);
  handleError(
    hipMemcpy(cylinderValidities.data(), devCylinderValidities, devCylinderValiditiesSize, hipMemcpyDeviceToHost));
  handleError(
    hipMemcpy(cellValues.data(), devCellValues, devCellValuesSize, hipMemcpyDeviceToHost));
  handleError(
    hipMemcpy(cellValidities.data(), devCellValidities, devCellValiditiesSize, hipMemcpyDeviceToHost));

  hipFree(devMinutiae);
  hipFree(devArea);
  hipFree(devCylinderValidities);
  hipFree(devCellValues);
  hipFree(devCellValidities);
}
