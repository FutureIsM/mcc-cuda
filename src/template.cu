#include "hip/hip_runtime.h"
#include <vector>

#include "errors.h"
#include "minutia.cuh"
#include "constants.cuh"

using namespace std;

// TODO: use global constant to precompute
__host__ __device__ __inline__
float gaussian(float value) {
  const float GS_DIV = sqrtf(2*M_PI) * SIGMA_S;
  const int expDenom = 2 * SIGMA_S * SIGMA_S;
  return expf(-(value*value)/expDenom) / GS_DIV;
}

__host__ __device__ __inline__
int sqrDistance(int x1, int y1, int x2, int y2) {
  int dx = x1 - x2;
  int dy = y1 - y2;
  return dx*dx + dy*dy;
}

__host__ __device__ __inline__
float distance(int x1, int y1, int x2, int y2) {
  return sqrtf(sqrDistance(x1, y1, x2, y2));
}

__host__ __device__ __inline__
float spatialContribution(
    int mt_x, int mt_y, int pi, int pj) {
  return gaussian(distance(mt_x, mt_y, pi, pj));
}

__host__ __device__ __inline__
float angle(float theta1, float theta2) {
  float diff = theta1-theta2;
  if (diff < -M_PI)
    return 2*M_PI + diff;
  if (diff >= M_PI)
    return -2*M_PI + diff;
  return diff;
}

// TODO: use gaussian function
__host__ __device__ __inline__
float directionalContribution(
    float m_theta, float mt_theta, float dphik) {
  return angle(dphik, angle(m_theta, mt_theta));
}

__global__
void buildCylinder(
    Minutia *minutiae, char *validArea,
    int rows, int cols, char2 *cells) {
  extern __shared__ int shared[];

  const int N = gridDim.x;
  Minutia *sharedMinutiae = (Minutia*)shared;
  char *contributed = (char*)&sharedMinutiae[N];

  if (blockIdx.x < N) {
    sharedMinutiae[blockIdx.x] = minutiae[blockIdx.x];
    contributed[blockIdx.x] = 0;
  }
  __syncthreads();

  int idxMinutia = blockIdx.x;
  Minutia *m = &sharedMinutiae[idxMinutia];

  int halfNS = -1 + (NS + 1) >> 1;
  int halfNSi = m->x - halfNS;
  int halfNSj = m->y - halfNS;

  float sint, cost;
  sincosf(m->theta, &sint, &cost);
  int pi = m->x + DELTA_S * (cost * halfNSi + sint * halfNSj);
  int pj = m->y + DELTA_S * (-sint * halfNSi + cost * halfNSj);

  const int SQR_SIGMA_S = 9 * SIGMA_S * SIGMA_S;

  char validity = pi >= 0 && pi < rows && pj >= 0 && pj < cols &&
    validArea[pi * cols + pj] && sqrDistance(m->x, m->y, pi, pj) <= R2;

  int idx = idxMinutia * NC + threadIdx.x * NS * NS + threadIdx.y * NS;
  for (int k = 0; k < ND; ++k, ++idx) {
    char value = 0;

    if (validity) {
      float dphik = -M_PI + (k + 0.5f) * DELTA_D;
      float sum = 0.0f;

      for (int l = 0; l < N; ++l) {
        if (l == idxMinutia)
          continue;

        Minutia *mt = &sharedMinutiae[l];
        if (sqrDistance(m->x, m->y, mt->x, mt->y) > SQR_SIGMA_S)
          continue;

        contributed[l] = 1;
        float sContrib = spatialContribution(mt->x, mt->y, pi, pj);
        float dContrib = directionalContribution(m->theta, mt->theta, dphik);
        sum += sContrib * dContrib;
      }

      if (sum >= MU_PSI)
        value = 1;
    }
    cells[idx] = make_char2(validity, value);
  }
  __syncthreads();

  // TODO: check cylinder validity
  if (threadIdx.x == 0 && threadIdx.y == 0) {
    int sum = 0;
    for (int i = 0; i < N; ++i)
      sum += contributed[i];
  }
}

__host__
vector<char2> buildTemplate(
    const vector<Minutia>& minutiae,
    const vector<char>& validArea,
    int rows, int cols) {
  Minutia *devMinutiae;
  char *devArea;
  char2 *devCells;
  size_t devMinutiaeSize = minutiae.size() * sizeof(Minutia);
  size_t devAreaSize = rows * cols * sizeof(char);
  size_t devCellsSize = minutiae.size() * NC * sizeof(char2);
  handleError(
    hipMalloc(&devMinutiae, devMinutiaeSize));
  handleError(
    hipMemcpy(devMinutiae, minutiae.data(), devMinutiaeSize, hipMemcpyHostToDevice));
  handleError(
    hipMalloc(&devArea, devAreaSize));
  handleError(
    hipMemcpy(devArea, validArea.data(), devAreaSize, hipMemcpyHostToDevice));
  handleError(
    hipMalloc(&devCells, devCellsSize));

  dim3 blockDim(NS, NS);
  int sharedSize = devMinutiaeSize + minutiae.size() * sizeof(char);
  buildCylinder<<<minutiae.size(), blockDim, sharedSize>>>(
    devMinutiae, devArea, rows, cols, devCells);

  vector<char2> ret(minutiae.size() * NC);
  handleError(
    hipMemcpy(ret.data(), devCells, devCellsSize, hipMemcpyDeviceToHost));

  hipFree(devMinutiae);
  hipFree(devArea);
  hipFree(devCells);

  return ret;
}
