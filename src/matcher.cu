#include "hip/hip_runtime.h"
#include "matcher.cuh"
#include "constants.cuh"
#include "errors.h"

using namespace std;

__global__
void binarizedTemplate(
    char *cellValidities,
    char *cellValues,
    int *binarizedValidities,
    int *binarizedValues) {
  int idxMinutia = blockIdx.x;
  int idxInt = threadIdx.x;
  int bits = sizeof(int) * 8;
  int intPerCylinder = NC / bits;
  int idx = idxMinutia * intPerCylinder + idxInt;
  int idxBit = idxMinutia * NC + idxInt * bits;

  int validity = 0, value = 0;
  for (int i = 0; i < bits; ++i) {
    validity |= cellValidities[idxBit+i] << i;
    value |= cellValues[idxBit+i] << i;
  }
  binarizedValidities[idx] = validity;
  binarizedValues[idx] = value;
}

__host__
float matchTemplate(
    const vector<char>& cylinderValidities1,
    const vector<char>& cellValidities1,
    const vector<char>& cellValues1,
    const vector<char>& cylinderValidities2,
    const vector<char>& cellValidities2,
    const vector<char>& cellValues2) {
  char *devCylinderValidities1, *devCylinderValidities2;
  char *devCellValidities1, *devCellValidities2;
  char *devCellValues1, *devCellValues2;
  size_t devCylinderValidities1Size = cylinderValidities1.size() * sizeof(char);
  size_t devCylinderValidities2Size = cylinderValidities2.size() * sizeof(char);
  size_t devCellValidities1Size = cellValidities1.size()  * sizeof(char);
  size_t devCellValidities2Size = cellValidities2.size()  * sizeof(char);
  size_t devCellValues1Size = cellValues1.size() * sizeof(char);
  size_t devCellValues2Size = cellValues2.size() * sizeof(char);
  handleError(
    hipMalloc(&devCylinderValidities1, devCylinderValidities1Size));
  handleError(
    hipMemcpy(devCylinderValidities1, cylinderValidities1.data(), devCylinderValidities1Size, hipMemcpyHostToDevice));
  handleError(
    hipMalloc(&devCylinderValidities2, devCylinderValidities2Size));
  handleError(
    hipMemcpy(devCylinderValidities2, cylinderValidities2.data(), devCylinderValidities2Size, hipMemcpyHostToDevice));
  handleError(
    hipMalloc(&devCellValidities1, devCellValidities1Size));
  handleError(
    hipMemcpy(devCellValidities1, cellValidities1.data(), devCellValidities1Size, hipMemcpyHostToDevice));
  handleError(
    hipMalloc(&devCellValidities2, devCellValidities2Size));
  handleError(
    hipMemcpy(devCellValidities2, cellValidities2.data(), devCellValidities2Size, hipMemcpyHostToDevice));
  handleError(
    hipMalloc(&devCellValues1, devCellValues1Size));
  handleError(
    hipMemcpy(devCellValues1, cellValues1.data(), devCellValues1Size, hipMemcpyHostToDevice));
  handleError(
    hipMalloc(&devCellValues2, devCellValues2Size));
  handleError(
    hipMemcpy(devCellValues2, cellValues2.data(), devCellValues2Size, hipMemcpyHostToDevice));

  int bits = sizeof(int) * 8;
  int *devBinarizedValidities1, *devBinarizedValues1;
  int *devBinarizedValidities2, *devBinarizedValues2;
  size_t devBinarizedValidities1Size = (cellValidities1.size() / bits) * sizeof(int);
  size_t devBinarizedValidities2Size = (cellValidities2.size() / bits) * sizeof(int);
  size_t devBinarizedValues1Size = (cellValues1.size() / bits) * sizeof(int);
  size_t devBinarizedValues2Size = (cellValues2.size() / bits) * sizeof(int);
  handleError(
    hipMalloc(&devBinarizedValidities1, devBinarizedValidities1Size));
  handleError(
    hipMalloc(&devBinarizedValidities2, devBinarizedValidities2Size));
  handleError(
    hipMalloc(&devBinarizedValues1, devBinarizedValues1Size));
  handleError(
    hipMalloc(&devBinarizedValues2, devBinarizedValues2Size));

  binarizedTemplate<<<cylinderValidities1.size(), NC/bits>>>(
    devCellValidities1, devCellValues1, devBinarizedValidities1, devBinarizedValues1);
  binarizedTemplate<<<cylinderValidities2.size(), NC/bits>>>(
    devCellValidities2, devCellValues2, devBinarizedValidities2, devBinarizedValues2);

  // TODO

  hipFree(devCylinderValidities1);
  hipFree(devCylinderValidities2);
  hipFree(devCellValidities1);
  hipFree(devCellValidities2);
  hipFree(devCellValues1);
  hipFree(devCellValues2);
  hipFree(devBinarizedValidities1);
  hipFree(devBinarizedValidities2);
  hipFree(devBinarizedValues1);
  hipFree(devBinarizedValues2);
}
