#include "hip/hip_runtime.h"
#include "matcher.cuh"
#include "constants.cuh"
#include "errors.h"
#include "debug.h"

#include <vector>
#include <algorithm>
#include <functional>

#define ceilMod(x,y) (x+y-1)/y

using namespace std;

__global__
void binarizedTemplate(
    char *cellValidities,
    char *cellValues,
    unsigned int *binarizedValidities,
    unsigned int *binarizedValues) {
  int idxMinutia = blockIdx.x;
  int idxInt = threadIdx.x;
  int intPerCylinder = NC / BITS;
  int idx = idxMinutia * intPerCylinder + idxInt;
  int idxBit = idxMinutia * NC + idxInt * BITS;

  unsigned int validity = 0, value = 0;
  for (int i = 0; i < BITS; ++i) {
    validity <<= 1U;
    validity |= cellValidities[idxBit+i];
    value <<= 1U;
    value |= cellValues[idxBit+i];
  }
  binarizedValidities[idx] = validity;
  binarizedValues[idx] = value;
}

__global__
void computeSimilarity(
    unsigned int *binarizedValidities1,
    unsigned int *binarizedValues1,
    unsigned int *binarizedValidities2,
    unsigned int *binarizedValues2,
    float *matrix, int rows, int cols) {
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  if (row >= rows || col >= cols) return;

  int intPerCylinder = NC/BITS;
  int rowIdx = row * intPerCylinder;
  int colIdx = col * intPerCylinder;

  unsigned int matchable = 0;
  int rowBits = 0, colBits = 0, xorBits = 0;

  for (int i = 0; i < intPerCylinder; ++i) {
    auto mask = binarizedValidities1[rowIdx+i] & binarizedValidities2[colIdx+i];
    auto rowValue = binarizedValues1[rowIdx+i] & mask;
    auto colValue = binarizedValues2[colIdx+i] & mask;
    auto xorValue = rowValue ^ colValue;
    matchable += mask;
    rowBits += __popc(rowValue);
    colBits += __popc(colValue);
    xorBits += __popc(xorValue);
  }

  float similarity = matchable && (rowBits+colBits)
    ? (1.0f - sqrtf(xorBits) / (sqrtf(rowBits)+sqrtf(colBits)))
    : 0.0f;
  matrix[row*cols + col] = similarity;
}

__host__
float LSS(const vector<float>& _matrix, int rows, int cols) {
  auto matrix(_matrix);
  auto sigmoid = [&](int value, float tau, float mu) {
    return 1.0f / (1.0f + expf(-tau * (value-mu)));
  };
  int n = MIN_NP + roundf(sigmoid(min(rows, cols), TAU_P, MU_P) * (MAX_NP - MIN_NP));
  debug("NP: %d\n", n);
  nth_element(matrix.begin(), matrix.begin()+n, matrix.end(), greater<float>());
  float sum = 0.0f;
  for (int i = 0; i < n; ++i)
    sum += matrix[i];
  return sum / n;
}

__host__
float matchTemplate(
    const vector<char>& cylinderValidities1,
    const vector<char>& cellValidities1,
    const vector<char>& cellValues1,
    const vector<char>& cylinderValidities2,
    const vector<char>& cellValidities2,
    const vector<char>& cellValues2,
    vector<float>& matrix) {

  int rows = cylinderValidities1.size();
  int cols = cylinderValidities2.size();

  char *devCylinderValidities1, *devCylinderValidities2;
  char *devCellValidities1, *devCellValidities2;
  char *devCellValues1, *devCellValues2;
  size_t devCylinderValidities1Size = cylinderValidities1.size() * sizeof(char);
  size_t devCylinderValidities2Size = cylinderValidities2.size() * sizeof(char);
  size_t devCellValidities1Size = cellValidities1.size()  * sizeof(char);
  size_t devCellValidities2Size = cellValidities2.size()  * sizeof(char);
  size_t devCellValues1Size = cellValues1.size() * sizeof(char);
  size_t devCellValues2Size = cellValues2.size() * sizeof(char);
  handleError(
    hipMalloc(&devCylinderValidities1, devCylinderValidities1Size));
  handleError(
    hipMemcpy(devCylinderValidities1, cylinderValidities1.data(), devCylinderValidities1Size, hipMemcpyHostToDevice));
  handleError(
    hipMalloc(&devCylinderValidities2, devCylinderValidities2Size));
  handleError(
    hipMemcpy(devCylinderValidities2, cylinderValidities2.data(), devCylinderValidities2Size, hipMemcpyHostToDevice));
  handleError(
    hipMalloc(&devCellValidities1, devCellValidities1Size));
  handleError(
    hipMemcpy(devCellValidities1, cellValidities1.data(), devCellValidities1Size, hipMemcpyHostToDevice));
  handleError(
    hipMalloc(&devCellValidities2, devCellValidities2Size));
  handleError(
    hipMemcpy(devCellValidities2, cellValidities2.data(), devCellValidities2Size, hipMemcpyHostToDevice));
  handleError(
    hipMalloc(&devCellValues1, devCellValues1Size));
  handleError(
    hipMemcpy(devCellValues1, cellValues1.data(), devCellValues1Size, hipMemcpyHostToDevice));
  handleError(
    hipMalloc(&devCellValues2, devCellValues2Size));
  handleError(
    hipMemcpy(devCellValues2, cellValues2.data(), devCellValues2Size, hipMemcpyHostToDevice));

  unsigned int *devBinarizedValidities1, *devBinarizedValues1;
  unsigned int *devBinarizedValidities2, *devBinarizedValues2;
  size_t devBinarizedValidities1Size = (cellValidities1.size()/BITS) * sizeof(unsigned int);
  size_t devBinarizedValidities2Size = (cellValidities2.size()/BITS) * sizeof(unsigned int);
  size_t devBinarizedValues1Size = (cellValues1.size()/BITS) * sizeof(unsigned int);
  size_t devBinarizedValues2Size = (cellValues2.size()/BITS) * sizeof(unsigned int);
  handleError(
    hipMalloc(&devBinarizedValidities1, devBinarizedValidities1Size));
  handleError(
    hipMalloc(&devBinarizedValidities2, devBinarizedValidities2Size));
  handleError(
    hipMalloc(&devBinarizedValues1, devBinarizedValues1Size));
  handleError(
    hipMalloc(&devBinarizedValues2, devBinarizedValues2Size));

  binarizedTemplate<<<cylinderValidities1.size(), NC/BITS>>>(
    devCellValidities1, devCellValues1, devBinarizedValidities1, devBinarizedValues1);
  handleError(
    hipPeekAtLastError());
  binarizedTemplate<<<cylinderValidities2.size(), NC/BITS>>>(
    devCellValidities2, devCellValues2, devBinarizedValidities2, devBinarizedValues2);
  handleError(
    hipPeekAtLastError());

  float *devMatrix;
  size_t devMatrixSize = rows * cols * sizeof(float);
  handleError(
    hipMalloc(&devMatrix, devMatrixSize));

  int threadPerDim = 32;
  dim3 blockCount(ceilMod(rows, threadPerDim), ceilMod(cols, threadPerDim));
  dim3 threadCount(threadPerDim, threadPerDim);
  computeSimilarity<<<blockCount, threadCount>>>(
    devBinarizedValidities1, devBinarizedValues1,
    devBinarizedValidities2, devBinarizedValues2,
    devMatrix, rows, cols);
  handleError(
    hipPeekAtLastError());

  matrix.resize(rows*cols);
  handleError(
    hipMemcpy(matrix.data(), devMatrix, devMatrixSize, hipMemcpyDeviceToHost));

  hipFree(devCylinderValidities1);
  hipFree(devCylinderValidities2);
  hipFree(devCellValidities1);
  hipFree(devCellValidities2);
  hipFree(devCellValues1);
  hipFree(devCellValues2);
  hipFree(devBinarizedValidities1);
  hipFree(devBinarizedValidities2);
  hipFree(devBinarizedValues1);
  hipFree(devBinarizedValues2);
  hipFree(devMatrix);

  return LSS(matrix, rows, cols);
}
