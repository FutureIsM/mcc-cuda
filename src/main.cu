#include <fstream>
#include <vector>
#include <iostream>

#include "errors.h"
#include "template.cuh"
#include "constants.cuh"
#include "area.cuh"

using namespace std;

int main() {
  int rows, cols, dpi, n;
  ifstream stream("data/1_1.txt");
  stream >> rows >> cols >> dpi >> n;
  vector<Minutia> minutiae;
  for (int i = 0; i < n; ++i) {
    int x, y;
    float theta;
    stream >> x >> y >> theta;
    minutiae.emplace_back(x, y, theta);
  }

  vector<char> x = buildValidArea(minutiae, rows, cols);
  handleError(
    hipDeviceSynchronize());
  return 0;
}
