#include <cstdlib>
#include <cstdio>
#include <fstream>
#include <vector>
#include <iostream>

#include "errors.h"
#include "constants.cuh"
#include "area.cuh"
#include "template.cuh"

using namespace std;

int main() {
  int rows, cols, dpi, n;
  ifstream stream("data/1_1.txt");
  stream >> rows >> cols >> dpi >> n;
  vector<Minutia> minutiae;
  for (int i = 0; i < n; ++i) {
    int x, y;
    float theta;
    stream >> x >> y >> theta;
    minutiae.emplace_back(x, y, theta);
  }

  auto area = buildValidArea(minutiae, rows, cols);
  auto t = buildTemplate(minutiae, area, rows, cols);
  handleError(hipDeviceSynchronize());
  return 0;
}
