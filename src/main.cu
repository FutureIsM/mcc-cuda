#include <cstdlib>
#include <cstdio>
#include <fstream>
#include <vector>
#include <iostream>

#include "errors.h"
#include "constants.cuh"
#include "area.cuh"
#include "template.cuh"

using namespace std;

int main() {
  int rows, cols, dpi, n;
  ifstream stream("data/1_1.txt");
  stream >> rows >> cols >> dpi >> n;
  vector<Minutia> minutiae;
  for (int i = 0; i < n; ++i) {
    int x, y;
    float theta;
    stream >> x >> y >> theta;
    minutiae.emplace_back(x, y, theta);
  }

  auto area = buildValidArea(minutiae, rows, cols);
  vector<char> values, validities;
  buildTemplate(minutiae, area, rows, cols, values, validities);
  handleError(hipDeviceSynchronize());  
  return 0;
}
