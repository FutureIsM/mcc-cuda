#include <chrono>
#include <iostream>

#include "mcc.cuh"
#include "constants.cuh"
#include "minutia.cuh"
#include "io.cuh"
#include "area.cuh"
#include "template.cuh"
#include "binarization.cuh"
#include "matcher.cuh"
#include "consolidation.cuh"
#include "errors.h"

using namespace std;

MCC::MCC(const char *input):
  input(input), loaded(false), built(false) {
}

MCC::~MCC() {
}

bool MCC::load() {
  if (loaded) return true;
  return loaded = loadMinutiaeFromFile(input, width, height, dpi, n, minutiae);
}

bool MCC::build() {
  if (built) return true;
  if (!loaded) return false;

  size_t devMinutiaeSize = minutiae.size() * sizeof(Minutia);
  handleError(
    hipMalloc(&devMinutiae, devMinutiaeSize));
  handleError(
    hipMemcpy(devMinutiae, minutiae.data(), devMinutiaeSize, hipMemcpyHostToDevice));

  size_t devAreaSize = width * height * sizeof(char);
  handleError(
    hipMalloc(&devArea, devAreaSize));

  size_t devCylinderValiditiesSize = minutiae.size() * sizeof(char);
  handleError(
    hipMalloc(&devCylinderValidities, devCylinderValiditiesSize));

  size_t devCellSize = minutiae.size() * NC * sizeof(char);
  handleError(
    hipMalloc(&devCellValidities, devCellSize));
  handleError(
    hipMalloc(&devCellValues, devCellSize));

  int intPerCylinder = NC/BITS;
  size_t devBinarizedSize = minutiae.size() * intPerCylinder * sizeof(unsigned int);
  handleError(
    hipMalloc(&devBinarizedValidities, devBinarizedSize));
  handleError(
    hipMalloc(&devBinarizedValues, devBinarizedSize));

  size_t devMatrixSize = MAX_MINUTIAE * MAX_MINUTIAE * sizeof(float);
  handleError(
    hipMalloc(&devMatrix, devMatrixSize));

  auto begin = std::chrono::high_resolution_clock::now();
  devBuildValidArea(minutiae, width, height, devArea);
  devBuildTemplate(
    devMinutiae, minutiae.size(),
    devArea, width, height,
    devCylinderValidities,
    devCellValidities,
    devCellValues);
  devBinarizedTemplate(minutiae.size(),
    devCellValidities, devCellValues,
    devBinarizedValidities, devBinarizedValues);
  handleError(
    hipDeviceSynchronize());
  auto end = chrono::high_resolution_clock::now();
  auto duration = chrono::duration_cast<chrono::microseconds>(end-begin).count();
  cout << "Time taken to build template: " << duration << " microseconds\n";

  return built = true;
}

void MCC::dispose() {
  if (!built) return;

  hipFree(devMinutiae);
  hipFree(devArea);
  hipFree(devCylinderValidities);
  hipFree(devCellValidities);
  hipFree(devCellValues);
  hipFree(devBinarizedValidities);
  hipFree(devBinarizedValues);
  hipFree(devMatrix);

  built = false;
}

bool MCC::match(const char *target,
    float &similarity, int &n, int &m, vector<float> &matrix) {
  MCC mcc(target);
  if (!mcc.load() || !mcc.build()) return false;

  n = minutiae.size();
  m = mcc.minutiae.size();

  auto begin = std::chrono::high_resolution_clock::now();
  devMatchTemplate(
    devMinutiae, n,
    devCylinderValidities, devBinarizedValidities, devBinarizedValues,
    mcc.devMinutiae, m,
    mcc.devCylinderValidities, mcc.devBinarizedValidities, mcc.devBinarizedValues,
    devMatrix);
  handleError(
    hipDeviceSynchronize());
  auto end = chrono::high_resolution_clock::now();
  auto duration = chrono::duration_cast<chrono::microseconds>(end-begin).count();
  cout << "Time taken to match templates: " << duration << " microseconds\n";

  size_t devMatrixSize = n * m * sizeof(float);
  matrix.resize(n * m);
  handleError(
    hipMemcpy(matrix.data(), devMatrix, devMatrixSize, hipMemcpyDeviceToHost));

  begin = std::chrono::high_resolution_clock::now();
  similarity = LSSR(matrix, n, m, minutiae, mcc.minutiae);
  end = chrono::high_resolution_clock::now();
  duration = chrono::duration_cast<chrono::microseconds>(end-begin).count();
  cout << "Time taken to compute global score: " << duration << " microseconds\n";

  mcc.dispose();
  return true;
}
