#include "mcc.cuh"
#include "constants.cuh"
#include "minutia.cuh"
#include "io.cuh"
#include "area.cuh"
#include "template.cuh"
#include "matcher.cuh"
#include "errors.h"

using namespace std;

MCC::MCC(const char *input):
  input(input), loaded(false), built(false) {
}

MCC::~MCC() {
}

bool MCC::load() {
  if (loaded) return true;
  return loaded = loadMinutiaeFromFile(input, &width, &height, &dpi, &n, minutiae);
}

bool MCC::build() {
  if (built) return true;
  if (!loaded) return false;

  size_t devMinutiaeSize = minutiae.size() * sizeof(Minutia);
  handleError(
    hipMalloc(&devMinutiae, devMinutiaeSize));
  handleError(
    hipMemcpy(devMinutiae, minutiae.data(), devMinutiaeSize, hipMemcpyHostToDevice));

  size_t devAreaSize = width * height * sizeof(char);
  handleError(
    hipMalloc(&devArea, devAreaSize));

  size_t devCylinderValiditiesSize = minutiae.size() * sizeof(char);
  handleError(
    hipMalloc(&devCylinderValidities, devCylinderValiditiesSize));

  size_t devCellValiditiesSize = minutiae.size() * NC * sizeof(char);
  handleError(
    hipMalloc(&devCellValidities, devCellValiditiesSize));

  size_t devCellValuesSize = minutiae.size() * NC * sizeof(char);
  handleError(
    hipMalloc(&devCellValues, devCellValuesSize));

  devBuildValidArea(minutiae, width, height, devArea);
  devBuildTemplate(
    devMinutiae, minutiae.size(),
    devArea, width, height,
    devCylinderValidities,
    devCellValidities,
    devCellValues);

  return built = true;
}

void MCC::dispose() {
  if (!built) return;

  hipFree(devMinutiae);
  hipFree(devArea);
  hipFree(devCylinderValidities);
  hipFree(devCellValidities);
  hipFree(devCellValues);

  built = false;
}

bool MCC::match(const char *target,
    float &similarity, int &n, int &m, vector<float> &matrix) {
  MCC mcc(target);
  if (!mcc.load() || !mcc.build()) return false;

  similarity = devMatchTemplate(
    devMinutiae, minutiae.size(),
    devCylinderValidities, devCellValidities, devCellValues,
    mcc.devMinutiae, mcc.minutiae.size(),
    mcc.devCylinderValidities, mcc.devCellValidities, mcc.devCellValues,
    matrix);
  n = minutiae.size();
  m = mcc.minutiae.size();

  mcc.dispose();
  return true;
}
